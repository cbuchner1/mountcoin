#include "hip/hip_runtime.h"
typedef unsigned char uint8_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

#define AES_WITHOUT_SHARED 0

// 0 is good for SM 35
// 1 is good for SM 30
#define DATA_LAYOUT_SWITCH 0

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

uint32_t *d_keccakOutputData[8];
uint32_t *d_keccakOutputExpandedAESKey1[8];
uint32_t *d_keccakOutputExpandedAESKey2[8];
uint32_t *d_hashOutput[8];
uint32_t *d_prepMemory[8];
uint32_t *d_abInput[8];

#define SPH_C64(x)    ((uint64_t)(x ## ULL))
#define SPH_C32(x)    ((uint32_t)(x ## U))

#include "cuda_helper.h"
////////////// TABLES FROM SPH

/* AES Helper for inline-usage from SPH */
#define AESx(x) SPH_C32(x)

static const uint32_t h_AES0[256] = {
	AESx(0xA56363C6), AESx(0x847C7CF8), AESx(0x997777EE), AESx(0x8D7B7BF6),
	AESx(0x0DF2F2FF), AESx(0xBD6B6BD6), AESx(0xB16F6FDE), AESx(0x54C5C591),
	AESx(0x50303060), AESx(0x03010102), AESx(0xA96767CE), AESx(0x7D2B2B56),
	AESx(0x19FEFEE7), AESx(0x62D7D7B5), AESx(0xE6ABAB4D), AESx(0x9A7676EC),
	AESx(0x45CACA8F), AESx(0x9D82821F), AESx(0x40C9C989), AESx(0x877D7DFA),
	AESx(0x15FAFAEF), AESx(0xEB5959B2), AESx(0xC947478E), AESx(0x0BF0F0FB),
	AESx(0xECADAD41), AESx(0x67D4D4B3), AESx(0xFDA2A25F), AESx(0xEAAFAF45),
	AESx(0xBF9C9C23), AESx(0xF7A4A453), AESx(0x967272E4), AESx(0x5BC0C09B),
	AESx(0xC2B7B775), AESx(0x1CFDFDE1), AESx(0xAE93933D), AESx(0x6A26264C),
	AESx(0x5A36366C), AESx(0x413F3F7E), AESx(0x02F7F7F5), AESx(0x4FCCCC83),
	AESx(0x5C343468), AESx(0xF4A5A551), AESx(0x34E5E5D1), AESx(0x08F1F1F9),
	AESx(0x937171E2), AESx(0x73D8D8AB), AESx(0x53313162), AESx(0x3F15152A),
	AESx(0x0C040408), AESx(0x52C7C795), AESx(0x65232346), AESx(0x5EC3C39D),
	AESx(0x28181830), AESx(0xA1969637), AESx(0x0F05050A), AESx(0xB59A9A2F),
	AESx(0x0907070E), AESx(0x36121224), AESx(0x9B80801B), AESx(0x3DE2E2DF),
	AESx(0x26EBEBCD), AESx(0x6927274E), AESx(0xCDB2B27F), AESx(0x9F7575EA),
	AESx(0x1B090912), AESx(0x9E83831D), AESx(0x742C2C58), AESx(0x2E1A1A34),
	AESx(0x2D1B1B36), AESx(0xB26E6EDC), AESx(0xEE5A5AB4), AESx(0xFBA0A05B),
	AESx(0xF65252A4), AESx(0x4D3B3B76), AESx(0x61D6D6B7), AESx(0xCEB3B37D),
	AESx(0x7B292952), AESx(0x3EE3E3DD), AESx(0x712F2F5E), AESx(0x97848413),
	AESx(0xF55353A6), AESx(0x68D1D1B9), AESx(0x00000000), AESx(0x2CEDEDC1),
	AESx(0x60202040), AESx(0x1FFCFCE3), AESx(0xC8B1B179), AESx(0xED5B5BB6),
	AESx(0xBE6A6AD4), AESx(0x46CBCB8D), AESx(0xD9BEBE67), AESx(0x4B393972),
	AESx(0xDE4A4A94), AESx(0xD44C4C98), AESx(0xE85858B0), AESx(0x4ACFCF85),
	AESx(0x6BD0D0BB), AESx(0x2AEFEFC5), AESx(0xE5AAAA4F), AESx(0x16FBFBED),
	AESx(0xC5434386), AESx(0xD74D4D9A), AESx(0x55333366), AESx(0x94858511),
	AESx(0xCF45458A), AESx(0x10F9F9E9), AESx(0x06020204), AESx(0x817F7FFE),
	AESx(0xF05050A0), AESx(0x443C3C78), AESx(0xBA9F9F25), AESx(0xE3A8A84B),
	AESx(0xF35151A2), AESx(0xFEA3A35D), AESx(0xC0404080), AESx(0x8A8F8F05),
	AESx(0xAD92923F), AESx(0xBC9D9D21), AESx(0x48383870), AESx(0x04F5F5F1),
	AESx(0xDFBCBC63), AESx(0xC1B6B677), AESx(0x75DADAAF), AESx(0x63212142),
	AESx(0x30101020), AESx(0x1AFFFFE5), AESx(0x0EF3F3FD), AESx(0x6DD2D2BF),
	AESx(0x4CCDCD81), AESx(0x140C0C18), AESx(0x35131326), AESx(0x2FECECC3),
	AESx(0xE15F5FBE), AESx(0xA2979735), AESx(0xCC444488), AESx(0x3917172E),
	AESx(0x57C4C493), AESx(0xF2A7A755), AESx(0x827E7EFC), AESx(0x473D3D7A),
	AESx(0xAC6464C8), AESx(0xE75D5DBA), AESx(0x2B191932), AESx(0x957373E6),
	AESx(0xA06060C0), AESx(0x98818119), AESx(0xD14F4F9E), AESx(0x7FDCDCA3),
	AESx(0x66222244), AESx(0x7E2A2A54), AESx(0xAB90903B), AESx(0x8388880B),
	AESx(0xCA46468C), AESx(0x29EEEEC7), AESx(0xD3B8B86B), AESx(0x3C141428),
	AESx(0x79DEDEA7), AESx(0xE25E5EBC), AESx(0x1D0B0B16), AESx(0x76DBDBAD),
	AESx(0x3BE0E0DB), AESx(0x56323264), AESx(0x4E3A3A74), AESx(0x1E0A0A14),
	AESx(0xDB494992), AESx(0x0A06060C), AESx(0x6C242448), AESx(0xE45C5CB8),
	AESx(0x5DC2C29F), AESx(0x6ED3D3BD), AESx(0xEFACAC43), AESx(0xA66262C4),
	AESx(0xA8919139), AESx(0xA4959531), AESx(0x37E4E4D3), AESx(0x8B7979F2),
	AESx(0x32E7E7D5), AESx(0x43C8C88B), AESx(0x5937376E), AESx(0xB76D6DDA),
	AESx(0x8C8D8D01), AESx(0x64D5D5B1), AESx(0xD24E4E9C), AESx(0xE0A9A949),
	AESx(0xB46C6CD8), AESx(0xFA5656AC), AESx(0x07F4F4F3), AESx(0x25EAEACF),
	AESx(0xAF6565CA), AESx(0x8E7A7AF4), AESx(0xE9AEAE47), AESx(0x18080810),
	AESx(0xD5BABA6F), AESx(0x887878F0), AESx(0x6F25254A), AESx(0x722E2E5C),
	AESx(0x241C1C38), AESx(0xF1A6A657), AESx(0xC7B4B473), AESx(0x51C6C697),
	AESx(0x23E8E8CB), AESx(0x7CDDDDA1), AESx(0x9C7474E8), AESx(0x211F1F3E),
	AESx(0xDD4B4B96), AESx(0xDCBDBD61), AESx(0x868B8B0D), AESx(0x858A8A0F),
	AESx(0x907070E0), AESx(0x423E3E7C), AESx(0xC4B5B571), AESx(0xAA6666CC),
	AESx(0xD8484890), AESx(0x05030306), AESx(0x01F6F6F7), AESx(0x120E0E1C),
	AESx(0xA36161C2), AESx(0x5F35356A), AESx(0xF95757AE), AESx(0xD0B9B969),
	AESx(0x91868617), AESx(0x58C1C199), AESx(0x271D1D3A), AESx(0xB99E9E27),
	AESx(0x38E1E1D9), AESx(0x13F8F8EB), AESx(0xB398982B), AESx(0x33111122),
	AESx(0xBB6969D2), AESx(0x70D9D9A9), AESx(0x898E8E07), AESx(0xA7949433),
	AESx(0xB69B9B2D), AESx(0x221E1E3C), AESx(0x92878715), AESx(0x20E9E9C9),
	AESx(0x49CECE87), AESx(0xFF5555AA), AESx(0x78282850), AESx(0x7ADFDFA5),
	AESx(0x8F8C8C03), AESx(0xF8A1A159), AESx(0x80898909), AESx(0x170D0D1A),
	AESx(0xDABFBF65), AESx(0x31E6E6D7), AESx(0xC6424284), AESx(0xB86868D0),
	AESx(0xC3414182), AESx(0xB0999929), AESx(0x772D2D5A), AESx(0x110F0F1E),
	AESx(0xCBB0B07B), AESx(0xFC5454A8), AESx(0xD6BBBB6D), AESx(0x3A16162C)
};

static const uint32_t h_AES1[256] = {
	AESx(0x6363C6A5), AESx(0x7C7CF884), AESx(0x7777EE99), AESx(0x7B7BF68D),
	AESx(0xF2F2FF0D), AESx(0x6B6BD6BD), AESx(0x6F6FDEB1), AESx(0xC5C59154),
	AESx(0x30306050), AESx(0x01010203), AESx(0x6767CEA9), AESx(0x2B2B567D),
	AESx(0xFEFEE719), AESx(0xD7D7B562), AESx(0xABAB4DE6), AESx(0x7676EC9A),
	AESx(0xCACA8F45), AESx(0x82821F9D), AESx(0xC9C98940), AESx(0x7D7DFA87),
	AESx(0xFAFAEF15), AESx(0x5959B2EB), AESx(0x47478EC9), AESx(0xF0F0FB0B),
	AESx(0xADAD41EC), AESx(0xD4D4B367), AESx(0xA2A25FFD), AESx(0xAFAF45EA),
	AESx(0x9C9C23BF), AESx(0xA4A453F7), AESx(0x7272E496), AESx(0xC0C09B5B),
	AESx(0xB7B775C2), AESx(0xFDFDE11C), AESx(0x93933DAE), AESx(0x26264C6A),
	AESx(0x36366C5A), AESx(0x3F3F7E41), AESx(0xF7F7F502), AESx(0xCCCC834F),
	AESx(0x3434685C), AESx(0xA5A551F4), AESx(0xE5E5D134), AESx(0xF1F1F908),
	AESx(0x7171E293), AESx(0xD8D8AB73), AESx(0x31316253), AESx(0x15152A3F),
	AESx(0x0404080C), AESx(0xC7C79552), AESx(0x23234665), AESx(0xC3C39D5E),
	AESx(0x18183028), AESx(0x969637A1), AESx(0x05050A0F), AESx(0x9A9A2FB5),
	AESx(0x07070E09), AESx(0x12122436), AESx(0x80801B9B), AESx(0xE2E2DF3D),
	AESx(0xEBEBCD26), AESx(0x27274E69), AESx(0xB2B27FCD), AESx(0x7575EA9F),
	AESx(0x0909121B), AESx(0x83831D9E), AESx(0x2C2C5874), AESx(0x1A1A342E),
	AESx(0x1B1B362D), AESx(0x6E6EDCB2), AESx(0x5A5AB4EE), AESx(0xA0A05BFB),
	AESx(0x5252A4F6), AESx(0x3B3B764D), AESx(0xD6D6B761), AESx(0xB3B37DCE),
	AESx(0x2929527B), AESx(0xE3E3DD3E), AESx(0x2F2F5E71), AESx(0x84841397),
	AESx(0x5353A6F5), AESx(0xD1D1B968), AESx(0x00000000), AESx(0xEDEDC12C),
	AESx(0x20204060), AESx(0xFCFCE31F), AESx(0xB1B179C8), AESx(0x5B5BB6ED),
	AESx(0x6A6AD4BE), AESx(0xCBCB8D46), AESx(0xBEBE67D9), AESx(0x3939724B),
	AESx(0x4A4A94DE), AESx(0x4C4C98D4), AESx(0x5858B0E8), AESx(0xCFCF854A),
	AESx(0xD0D0BB6B), AESx(0xEFEFC52A), AESx(0xAAAA4FE5), AESx(0xFBFBED16),
	AESx(0x434386C5), AESx(0x4D4D9AD7), AESx(0x33336655), AESx(0x85851194),
	AESx(0x45458ACF), AESx(0xF9F9E910), AESx(0x02020406), AESx(0x7F7FFE81),
	AESx(0x5050A0F0), AESx(0x3C3C7844), AESx(0x9F9F25BA), AESx(0xA8A84BE3),
	AESx(0x5151A2F3), AESx(0xA3A35DFE), AESx(0x404080C0), AESx(0x8F8F058A),
	AESx(0x92923FAD), AESx(0x9D9D21BC), AESx(0x38387048), AESx(0xF5F5F104),
	AESx(0xBCBC63DF), AESx(0xB6B677C1), AESx(0xDADAAF75), AESx(0x21214263),
	AESx(0x10102030), AESx(0xFFFFE51A), AESx(0xF3F3FD0E), AESx(0xD2D2BF6D),
	AESx(0xCDCD814C), AESx(0x0C0C1814), AESx(0x13132635), AESx(0xECECC32F),
	AESx(0x5F5FBEE1), AESx(0x979735A2), AESx(0x444488CC), AESx(0x17172E39),
	AESx(0xC4C49357), AESx(0xA7A755F2), AESx(0x7E7EFC82), AESx(0x3D3D7A47),
	AESx(0x6464C8AC), AESx(0x5D5DBAE7), AESx(0x1919322B), AESx(0x7373E695),
	AESx(0x6060C0A0), AESx(0x81811998), AESx(0x4F4F9ED1), AESx(0xDCDCA37F),
	AESx(0x22224466), AESx(0x2A2A547E), AESx(0x90903BAB), AESx(0x88880B83),
	AESx(0x46468CCA), AESx(0xEEEEC729), AESx(0xB8B86BD3), AESx(0x1414283C),
	AESx(0xDEDEA779), AESx(0x5E5EBCE2), AESx(0x0B0B161D), AESx(0xDBDBAD76),
	AESx(0xE0E0DB3B), AESx(0x32326456), AESx(0x3A3A744E), AESx(0x0A0A141E),
	AESx(0x494992DB), AESx(0x06060C0A), AESx(0x2424486C), AESx(0x5C5CB8E4),
	AESx(0xC2C29F5D), AESx(0xD3D3BD6E), AESx(0xACAC43EF), AESx(0x6262C4A6),
	AESx(0x919139A8), AESx(0x959531A4), AESx(0xE4E4D337), AESx(0x7979F28B),
	AESx(0xE7E7D532), AESx(0xC8C88B43), AESx(0x37376E59), AESx(0x6D6DDAB7),
	AESx(0x8D8D018C), AESx(0xD5D5B164), AESx(0x4E4E9CD2), AESx(0xA9A949E0),
	AESx(0x6C6CD8B4), AESx(0x5656ACFA), AESx(0xF4F4F307), AESx(0xEAEACF25),
	AESx(0x6565CAAF), AESx(0x7A7AF48E), AESx(0xAEAE47E9), AESx(0x08081018),
	AESx(0xBABA6FD5), AESx(0x7878F088), AESx(0x25254A6F), AESx(0x2E2E5C72),
	AESx(0x1C1C3824), AESx(0xA6A657F1), AESx(0xB4B473C7), AESx(0xC6C69751),
	AESx(0xE8E8CB23), AESx(0xDDDDA17C), AESx(0x7474E89C), AESx(0x1F1F3E21),
	AESx(0x4B4B96DD), AESx(0xBDBD61DC), AESx(0x8B8B0D86), AESx(0x8A8A0F85),
	AESx(0x7070E090), AESx(0x3E3E7C42), AESx(0xB5B571C4), AESx(0x6666CCAA),
	AESx(0x484890D8), AESx(0x03030605), AESx(0xF6F6F701), AESx(0x0E0E1C12),
	AESx(0x6161C2A3), AESx(0x35356A5F), AESx(0x5757AEF9), AESx(0xB9B969D0),
	AESx(0x86861791), AESx(0xC1C19958), AESx(0x1D1D3A27), AESx(0x9E9E27B9),
	AESx(0xE1E1D938), AESx(0xF8F8EB13), AESx(0x98982BB3), AESx(0x11112233),
	AESx(0x6969D2BB), AESx(0xD9D9A970), AESx(0x8E8E0789), AESx(0x949433A7),
	AESx(0x9B9B2DB6), AESx(0x1E1E3C22), AESx(0x87871592), AESx(0xE9E9C920),
	AESx(0xCECE8749), AESx(0x5555AAFF), AESx(0x28285078), AESx(0xDFDFA57A),
	AESx(0x8C8C038F), AESx(0xA1A159F8), AESx(0x89890980), AESx(0x0D0D1A17),
	AESx(0xBFBF65DA), AESx(0xE6E6D731), AESx(0x424284C6), AESx(0x6868D0B8),
	AESx(0x414182C3), AESx(0x999929B0), AESx(0x2D2D5A77), AESx(0x0F0F1E11),
	AESx(0xB0B07BCB), AESx(0x5454A8FC), AESx(0xBBBB6DD6), AESx(0x16162C3A)
};

static const uint32_t h_AES2[256] = {
	AESx(0x63C6A563), AESx(0x7CF8847C), AESx(0x77EE9977), AESx(0x7BF68D7B),
	AESx(0xF2FF0DF2), AESx(0x6BD6BD6B), AESx(0x6FDEB16F), AESx(0xC59154C5),
	AESx(0x30605030), AESx(0x01020301), AESx(0x67CEA967), AESx(0x2B567D2B),
	AESx(0xFEE719FE), AESx(0xD7B562D7), AESx(0xAB4DE6AB), AESx(0x76EC9A76),
	AESx(0xCA8F45CA), AESx(0x821F9D82), AESx(0xC98940C9), AESx(0x7DFA877D),
	AESx(0xFAEF15FA), AESx(0x59B2EB59), AESx(0x478EC947), AESx(0xF0FB0BF0),
	AESx(0xAD41ECAD), AESx(0xD4B367D4), AESx(0xA25FFDA2), AESx(0xAF45EAAF),
	AESx(0x9C23BF9C), AESx(0xA453F7A4), AESx(0x72E49672), AESx(0xC09B5BC0),
	AESx(0xB775C2B7), AESx(0xFDE11CFD), AESx(0x933DAE93), AESx(0x264C6A26),
	AESx(0x366C5A36), AESx(0x3F7E413F), AESx(0xF7F502F7), AESx(0xCC834FCC),
	AESx(0x34685C34), AESx(0xA551F4A5), AESx(0xE5D134E5), AESx(0xF1F908F1),
	AESx(0x71E29371), AESx(0xD8AB73D8), AESx(0x31625331), AESx(0x152A3F15),
	AESx(0x04080C04), AESx(0xC79552C7), AESx(0x23466523), AESx(0xC39D5EC3),
	AESx(0x18302818), AESx(0x9637A196), AESx(0x050A0F05), AESx(0x9A2FB59A),
	AESx(0x070E0907), AESx(0x12243612), AESx(0x801B9B80), AESx(0xE2DF3DE2),
	AESx(0xEBCD26EB), AESx(0x274E6927), AESx(0xB27FCDB2), AESx(0x75EA9F75),
	AESx(0x09121B09), AESx(0x831D9E83), AESx(0x2C58742C), AESx(0x1A342E1A),
	AESx(0x1B362D1B), AESx(0x6EDCB26E), AESx(0x5AB4EE5A), AESx(0xA05BFBA0),
	AESx(0x52A4F652), AESx(0x3B764D3B), AESx(0xD6B761D6), AESx(0xB37DCEB3),
	AESx(0x29527B29), AESx(0xE3DD3EE3), AESx(0x2F5E712F), AESx(0x84139784),
	AESx(0x53A6F553), AESx(0xD1B968D1), AESx(0x00000000), AESx(0xEDC12CED),
	AESx(0x20406020), AESx(0xFCE31FFC), AESx(0xB179C8B1), AESx(0x5BB6ED5B),
	AESx(0x6AD4BE6A), AESx(0xCB8D46CB), AESx(0xBE67D9BE), AESx(0x39724B39),
	AESx(0x4A94DE4A), AESx(0x4C98D44C), AESx(0x58B0E858), AESx(0xCF854ACF),
	AESx(0xD0BB6BD0), AESx(0xEFC52AEF), AESx(0xAA4FE5AA), AESx(0xFBED16FB),
	AESx(0x4386C543), AESx(0x4D9AD74D), AESx(0x33665533), AESx(0x85119485),
	AESx(0x458ACF45), AESx(0xF9E910F9), AESx(0x02040602), AESx(0x7FFE817F),
	AESx(0x50A0F050), AESx(0x3C78443C), AESx(0x9F25BA9F), AESx(0xA84BE3A8),
	AESx(0x51A2F351), AESx(0xA35DFEA3), AESx(0x4080C040), AESx(0x8F058A8F),
	AESx(0x923FAD92), AESx(0x9D21BC9D), AESx(0x38704838), AESx(0xF5F104F5),
	AESx(0xBC63DFBC), AESx(0xB677C1B6), AESx(0xDAAF75DA), AESx(0x21426321),
	AESx(0x10203010), AESx(0xFFE51AFF), AESx(0xF3FD0EF3), AESx(0xD2BF6DD2),
	AESx(0xCD814CCD), AESx(0x0C18140C), AESx(0x13263513), AESx(0xECC32FEC),
	AESx(0x5FBEE15F), AESx(0x9735A297), AESx(0x4488CC44), AESx(0x172E3917),
	AESx(0xC49357C4), AESx(0xA755F2A7), AESx(0x7EFC827E), AESx(0x3D7A473D),
	AESx(0x64C8AC64), AESx(0x5DBAE75D), AESx(0x19322B19), AESx(0x73E69573),
	AESx(0x60C0A060), AESx(0x81199881), AESx(0x4F9ED14F), AESx(0xDCA37FDC),
	AESx(0x22446622), AESx(0x2A547E2A), AESx(0x903BAB90), AESx(0x880B8388),
	AESx(0x468CCA46), AESx(0xEEC729EE), AESx(0xB86BD3B8), AESx(0x14283C14),
	AESx(0xDEA779DE), AESx(0x5EBCE25E), AESx(0x0B161D0B), AESx(0xDBAD76DB),
	AESx(0xE0DB3BE0), AESx(0x32645632), AESx(0x3A744E3A), AESx(0x0A141E0A),
	AESx(0x4992DB49), AESx(0x060C0A06), AESx(0x24486C24), AESx(0x5CB8E45C),
	AESx(0xC29F5DC2), AESx(0xD3BD6ED3), AESx(0xAC43EFAC), AESx(0x62C4A662),
	AESx(0x9139A891), AESx(0x9531A495), AESx(0xE4D337E4), AESx(0x79F28B79),
	AESx(0xE7D532E7), AESx(0xC88B43C8), AESx(0x376E5937), AESx(0x6DDAB76D),
	AESx(0x8D018C8D), AESx(0xD5B164D5), AESx(0x4E9CD24E), AESx(0xA949E0A9),
	AESx(0x6CD8B46C), AESx(0x56ACFA56), AESx(0xF4F307F4), AESx(0xEACF25EA),
	AESx(0x65CAAF65), AESx(0x7AF48E7A), AESx(0xAE47E9AE), AESx(0x08101808),
	AESx(0xBA6FD5BA), AESx(0x78F08878), AESx(0x254A6F25), AESx(0x2E5C722E),
	AESx(0x1C38241C), AESx(0xA657F1A6), AESx(0xB473C7B4), AESx(0xC69751C6),
	AESx(0xE8CB23E8), AESx(0xDDA17CDD), AESx(0x74E89C74), AESx(0x1F3E211F),
	AESx(0x4B96DD4B), AESx(0xBD61DCBD), AESx(0x8B0D868B), AESx(0x8A0F858A),
	AESx(0x70E09070), AESx(0x3E7C423E), AESx(0xB571C4B5), AESx(0x66CCAA66),
	AESx(0x4890D848), AESx(0x03060503), AESx(0xF6F701F6), AESx(0x0E1C120E),
	AESx(0x61C2A361), AESx(0x356A5F35), AESx(0x57AEF957), AESx(0xB969D0B9),
	AESx(0x86179186), AESx(0xC19958C1), AESx(0x1D3A271D), AESx(0x9E27B99E),
	AESx(0xE1D938E1), AESx(0xF8EB13F8), AESx(0x982BB398), AESx(0x11223311),
	AESx(0x69D2BB69), AESx(0xD9A970D9), AESx(0x8E07898E), AESx(0x9433A794),
	AESx(0x9B2DB69B), AESx(0x1E3C221E), AESx(0x87159287), AESx(0xE9C920E9),
	AESx(0xCE8749CE), AESx(0x55AAFF55), AESx(0x28507828), AESx(0xDFA57ADF),
	AESx(0x8C038F8C), AESx(0xA159F8A1), AESx(0x89098089), AESx(0x0D1A170D),
	AESx(0xBF65DABF), AESx(0xE6D731E6), AESx(0x4284C642), AESx(0x68D0B868),
	AESx(0x4182C341), AESx(0x9929B099), AESx(0x2D5A772D), AESx(0x0F1E110F),
	AESx(0xB07BCBB0), AESx(0x54A8FC54), AESx(0xBB6DD6BB), AESx(0x162C3A16)
};

static const uint32_t h_AES3[256] = {
	AESx(0xC6A56363), AESx(0xF8847C7C), AESx(0xEE997777), AESx(0xF68D7B7B),
	AESx(0xFF0DF2F2), AESx(0xD6BD6B6B), AESx(0xDEB16F6F), AESx(0x9154C5C5),
	AESx(0x60503030), AESx(0x02030101), AESx(0xCEA96767), AESx(0x567D2B2B),
	AESx(0xE719FEFE), AESx(0xB562D7D7), AESx(0x4DE6ABAB), AESx(0xEC9A7676),
	AESx(0x8F45CACA), AESx(0x1F9D8282), AESx(0x8940C9C9), AESx(0xFA877D7D),
	AESx(0xEF15FAFA), AESx(0xB2EB5959), AESx(0x8EC94747), AESx(0xFB0BF0F0),
	AESx(0x41ECADAD), AESx(0xB367D4D4), AESx(0x5FFDA2A2), AESx(0x45EAAFAF),
	AESx(0x23BF9C9C), AESx(0x53F7A4A4), AESx(0xE4967272), AESx(0x9B5BC0C0),
	AESx(0x75C2B7B7), AESx(0xE11CFDFD), AESx(0x3DAE9393), AESx(0x4C6A2626),
	AESx(0x6C5A3636), AESx(0x7E413F3F), AESx(0xF502F7F7), AESx(0x834FCCCC),
	AESx(0x685C3434), AESx(0x51F4A5A5), AESx(0xD134E5E5), AESx(0xF908F1F1),
	AESx(0xE2937171), AESx(0xAB73D8D8), AESx(0x62533131), AESx(0x2A3F1515),
	AESx(0x080C0404), AESx(0x9552C7C7), AESx(0x46652323), AESx(0x9D5EC3C3),
	AESx(0x30281818), AESx(0x37A19696), AESx(0x0A0F0505), AESx(0x2FB59A9A),
	AESx(0x0E090707), AESx(0x24361212), AESx(0x1B9B8080), AESx(0xDF3DE2E2),
	AESx(0xCD26EBEB), AESx(0x4E692727), AESx(0x7FCDB2B2), AESx(0xEA9F7575),
	AESx(0x121B0909), AESx(0x1D9E8383), AESx(0x58742C2C), AESx(0x342E1A1A),
	AESx(0x362D1B1B), AESx(0xDCB26E6E), AESx(0xB4EE5A5A), AESx(0x5BFBA0A0),
	AESx(0xA4F65252), AESx(0x764D3B3B), AESx(0xB761D6D6), AESx(0x7DCEB3B3),
	AESx(0x527B2929), AESx(0xDD3EE3E3), AESx(0x5E712F2F), AESx(0x13978484),
	AESx(0xA6F55353), AESx(0xB968D1D1), AESx(0x00000000), AESx(0xC12CEDED),
	AESx(0x40602020), AESx(0xE31FFCFC), AESx(0x79C8B1B1), AESx(0xB6ED5B5B),
	AESx(0xD4BE6A6A), AESx(0x8D46CBCB), AESx(0x67D9BEBE), AESx(0x724B3939),
	AESx(0x94DE4A4A), AESx(0x98D44C4C), AESx(0xB0E85858), AESx(0x854ACFCF),
	AESx(0xBB6BD0D0), AESx(0xC52AEFEF), AESx(0x4FE5AAAA), AESx(0xED16FBFB),
	AESx(0x86C54343), AESx(0x9AD74D4D), AESx(0x66553333), AESx(0x11948585),
	AESx(0x8ACF4545), AESx(0xE910F9F9), AESx(0x04060202), AESx(0xFE817F7F),
	AESx(0xA0F05050), AESx(0x78443C3C), AESx(0x25BA9F9F), AESx(0x4BE3A8A8),
	AESx(0xA2F35151), AESx(0x5DFEA3A3), AESx(0x80C04040), AESx(0x058A8F8F),
	AESx(0x3FAD9292), AESx(0x21BC9D9D), AESx(0x70483838), AESx(0xF104F5F5),
	AESx(0x63DFBCBC), AESx(0x77C1B6B6), AESx(0xAF75DADA), AESx(0x42632121),
	AESx(0x20301010), AESx(0xE51AFFFF), AESx(0xFD0EF3F3), AESx(0xBF6DD2D2),
	AESx(0x814CCDCD), AESx(0x18140C0C), AESx(0x26351313), AESx(0xC32FECEC),
	AESx(0xBEE15F5F), AESx(0x35A29797), AESx(0x88CC4444), AESx(0x2E391717),
	AESx(0x9357C4C4), AESx(0x55F2A7A7), AESx(0xFC827E7E), AESx(0x7A473D3D),
	AESx(0xC8AC6464), AESx(0xBAE75D5D), AESx(0x322B1919), AESx(0xE6957373),
	AESx(0xC0A06060), AESx(0x19988181), AESx(0x9ED14F4F), AESx(0xA37FDCDC),
	AESx(0x44662222), AESx(0x547E2A2A), AESx(0x3BAB9090), AESx(0x0B838888),
	AESx(0x8CCA4646), AESx(0xC729EEEE), AESx(0x6BD3B8B8), AESx(0x283C1414),
	AESx(0xA779DEDE), AESx(0xBCE25E5E), AESx(0x161D0B0B), AESx(0xAD76DBDB),
	AESx(0xDB3BE0E0), AESx(0x64563232), AESx(0x744E3A3A), AESx(0x141E0A0A),
	AESx(0x92DB4949), AESx(0x0C0A0606), AESx(0x486C2424), AESx(0xB8E45C5C),
	AESx(0x9F5DC2C2), AESx(0xBD6ED3D3), AESx(0x43EFACAC), AESx(0xC4A66262),
	AESx(0x39A89191), AESx(0x31A49595), AESx(0xD337E4E4), AESx(0xF28B7979),
	AESx(0xD532E7E7), AESx(0x8B43C8C8), AESx(0x6E593737), AESx(0xDAB76D6D),
	AESx(0x018C8D8D), AESx(0xB164D5D5), AESx(0x9CD24E4E), AESx(0x49E0A9A9),
	AESx(0xD8B46C6C), AESx(0xACFA5656), AESx(0xF307F4F4), AESx(0xCF25EAEA),
	AESx(0xCAAF6565), AESx(0xF48E7A7A), AESx(0x47E9AEAE), AESx(0x10180808),
	AESx(0x6FD5BABA), AESx(0xF0887878), AESx(0x4A6F2525), AESx(0x5C722E2E),
	AESx(0x38241C1C), AESx(0x57F1A6A6), AESx(0x73C7B4B4), AESx(0x9751C6C6),
	AESx(0xCB23E8E8), AESx(0xA17CDDDD), AESx(0xE89C7474), AESx(0x3E211F1F),
	AESx(0x96DD4B4B), AESx(0x61DCBDBD), AESx(0x0D868B8B), AESx(0x0F858A8A),
	AESx(0xE0907070), AESx(0x7C423E3E), AESx(0x71C4B5B5), AESx(0xCCAA6666),
	AESx(0x90D84848), AESx(0x06050303), AESx(0xF701F6F6), AESx(0x1C120E0E),
	AESx(0xC2A36161), AESx(0x6A5F3535), AESx(0xAEF95757), AESx(0x69D0B9B9),
	AESx(0x17918686), AESx(0x9958C1C1), AESx(0x3A271D1D), AESx(0x27B99E9E),
	AESx(0xD938E1E1), AESx(0xEB13F8F8), AESx(0x2BB39898), AESx(0x22331111),
	AESx(0xD2BB6969), AESx(0xA970D9D9), AESx(0x07898E8E), AESx(0x33A79494),
	AESx(0x2DB69B9B), AESx(0x3C221E1E), AESx(0x15928787), AESx(0xC920E9E9),
	AESx(0x8749CECE), AESx(0xAAFF5555), AESx(0x50782828), AESx(0xA57ADFDF),
	AESx(0x038F8C8C), AESx(0x59F8A1A1), AESx(0x09808989), AESx(0x1A170D0D),
	AESx(0x65DABFBF), AESx(0xD731E6E6), AESx(0x84C64242), AESx(0xD0B86868),
	AESx(0x82C34141), AESx(0x29B09999), AESx(0x5A772D2D), AESx(0x1E110F0F),
	AESx(0x7BCBB0B0), AESx(0xA8FC5454), AESx(0x6DD6BBBB), AESx(0x2C3A1616)
};

static __constant__ uint32_t d_AES0[256];
static __constant__ uint32_t d_AES1[256];
static __constant__ uint32_t d_AES2[256];
static __constant__ uint32_t d_AES3[256];

static void aes_cpu_init()
{
	hipMemcpyToSymbol(HIP_SYMBOL( d_AES0),
                        h_AES0,
                        sizeof(h_AES0),
                        0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL( d_AES1),
                        h_AES1,
                        sizeof(h_AES1),
                        0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL( d_AES2),
                        h_AES2,
                        sizeof(h_AES2),
                        0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL( d_AES3),
                        h_AES3,
                        sizeof(h_AES3),
                        0, hipMemcpyHostToDevice);
}

static __device__ __forceinline__ void aes_gpu_init(uint32_t *sharedMemory)
{
	if(threadIdx.x < 256)
	{
		sharedMemory[threadIdx.x] = d_AES0[threadIdx.x];
		sharedMemory[threadIdx.x+256] = d_AES1[threadIdx.x];
		sharedMemory[threadIdx.x+512] = d_AES2[threadIdx.x];
		sharedMemory[threadIdx.x+768] = d_AES3[threadIdx.x];
	}
}

static __device__ __forceinline__ void aes_gpu_init_flex(uint32_t *sharedMemory)
{
	if (blockDim.x >= 256)
	{
		aes_gpu_init(sharedMemory);
	}
	else
	{
		int chunksize = 1 << (31 - __clz(blockDim.x));
		if (threadIdx.x < chunksize)
		{
			int chunks = 256/chunksize;

			for(int i=0;i<chunks;i++)
			{
				int index = (threadIdx.x * chunks) + i;
				sharedMemory[index + 0] = d_AES0[index];
				sharedMemory[index+256] = d_AES1[index];
				sharedMemory[index+512] = d_AES2[index];
				sharedMemory[index+768] = d_AES3[index];
			}
		}
	}
}

static __device__ __forceinline__ void aes_round_quad(
	const uint32_t *sharedMemory,
	uint32_t x0, uint32_t x1, uint32_t x2, uint32_t x3,
	uint32_t k, uint32_t &y)
{
	uint32_t idx0, idx1, idx2, idx3;
	idx0 = __byte_perm(x0, 0, 0x4440);
	idx1 = __byte_perm(x1, 0, 0x4441) + 256;
	idx2 = __byte_perm(x2, 0, 0x4442) + 512;
	idx3 = __byte_perm(x3, 0, 0x4443) + 768;
	y =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3] ^ k;
}

static __device__ __forceinline__ void aes_round(
	const uint32_t *sharedMemory,
	uint32_t x0, uint32_t x1, uint32_t x2, uint32_t x3, 
	uint32_t k0, uint32_t k1, uint32_t k2, uint32_t k3, 
	uint32_t &y0, uint32_t &y1, uint32_t &y2, uint32_t &y3)
{
	uint32_t idx0, idx1, idx2, idx3;

	idx0 = __byte_perm(x0, 0, 0x4440);
	idx1 = __byte_perm(x1, 0, 0x4441) + 256;
	idx2 = __byte_perm(x2, 0, 0x4442) + 512;
	idx3 = __byte_perm(x3, 0, 0x4443) + 768;
	y0 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3] ^ k0;

	idx0 = __byte_perm(x1, 0, 0x4440);
	idx1 = __byte_perm(x2, 0, 0x4441) + 256;
	idx2 = __byte_perm(x3, 0, 0x4442) + 512;
	idx3 = __byte_perm(x0, 0, 0x4443) + 768;

	y1 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3] ^ k1;

	idx0 = __byte_perm(x2, 0, 0x4440);
	idx1 = __byte_perm(x3, 0, 0x4441) + 256;
	idx2 = __byte_perm(x0, 0, 0x4442) + 512;
	idx3 = __byte_perm(x1, 0, 0x4443) + 768;
	y2 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3] ^ k2;

	idx0 = __byte_perm(x3, 0, 0x4440);
	idx1 = __byte_perm(x0, 0, 0x4441) + 256;
	idx2 = __byte_perm(x1, 0, 0x4442) + 512;
	idx3 = __byte_perm(x2, 0, 0x4443) + 768;
	y3 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3] ^ k3;
}

static __device__ __forceinline__ void aes_round(
	const uint32_t *sharedMemory,
	uint32_t x0, uint32_t x1, uint32_t x2, uint32_t x3, 
	uint32_t k0, 
	uint32_t &y0, uint32_t &y1, uint32_t &y2, uint32_t &y3)
{
	uint32_t idx0, idx1, idx2, idx3;

	idx0 = __byte_perm(x0, 0, 0x4440);
	idx1 = __byte_perm(x1, 0, 0x4441) + 256;
	idx2 = __byte_perm(x2, 0, 0x4442) + 512;
	idx3 = __byte_perm(x3, 0, 0x4443) + 768;
	y0 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3] ^ 
		k0;

	idx0 = __byte_perm(x1, 0, 0x4440);
	idx1 = __byte_perm(x2, 0, 0x4441) + 256;
	idx2 = __byte_perm(x3, 0, 0x4442) + 512;
	idx3 = __byte_perm(x0, 0, 0x4443) + 768;

	y1 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3]; // ^k3

	idx0 = __byte_perm(x2, 0, 0x4440);
	idx1 = __byte_perm(x3, 0, 0x4441) + 256;
	idx2 = __byte_perm(x0, 0, 0x4442) + 512;
	idx3 = __byte_perm(x1, 0, 0x4443) + 768;
	y2 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3]; // ^k2

	idx0 = __byte_perm(x3, 0, 0x4440);
	idx1 = __byte_perm(x0, 0, 0x4441) + 256;
	idx2 = __byte_perm(x1, 0, 0x4442) + 512;
	idx3 = __byte_perm(x2, 0, 0x4443) + 768;
	y3 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3]; // ^k3
}

static __device__ __forceinline__ void aes_round(
	const uint32_t *sharedMemory,
	uint32_t x0, uint32_t x1, uint32_t x2, uint32_t x3, 
	uint32_t &y0, uint32_t &y1, uint32_t &y2, uint32_t &y3)
{
	uint32_t idx0, idx1, idx2, idx3;

	idx0 = __byte_perm(x0, 0, 0x4440);
	idx1 = __byte_perm(x1, 0, 0x4441) + 256;
	idx2 = __byte_perm(x2, 0, 0x4442) + 512;
	idx3 = __byte_perm(x3, 0, 0x4443) + 768;
	y0 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3];

	idx0 = __byte_perm(x1, 0, 0x4440);
	idx1 = __byte_perm(x2, 0, 0x4441) + 256;
	idx2 = __byte_perm(x3, 0, 0x4442) + 512;
	idx3 = __byte_perm(x0, 0, 0x4443) + 768;

	y1 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3]; // ^k3

	idx0 = __byte_perm(x2, 0, 0x4440);
	idx1 = __byte_perm(x3, 0, 0x4441) + 256;
	idx2 = __byte_perm(x0, 0, 0x4442) + 512;
	idx3 = __byte_perm(x1, 0, 0x4443) + 768;
	y2 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3]; // ^k2

	idx0 = __byte_perm(x3, 0, 0x4440);
	idx1 = __byte_perm(x0, 0, 0x4441) + 256;
	idx2 = __byte_perm(x1, 0, 0x4442) + 512;
	idx3 = __byte_perm(x2, 0, 0x4443) + 768;
	y3 =sharedMemory[idx0] ^ 
		sharedMemory[idx1] ^ 
		sharedMemory[idx2] ^ 
		sharedMemory[idx3]; // ^k3
}


///////////// AES ohne shared memory
static const uint32_t FSb[64] =
{
    0x7B777C63, 0xC56F6BF2, 0x2B670130, 0x76ABD7FE, 
	0x7DC982CA, 0xF04759FA, 0xAFA2D4AD, 0xC072A49C, 
	0x2693FDB7, 0xCCF73F36, 0xF1E5A534, 0x1531D871, 
	0xC323C704, 0x9A059618, 0xE2801207, 0x75B227EB, 
	0x1A2C8309, 0xA05A6E1B, 0xB3D63B52, 0x842FE329, 
	0xED00D153, 0x5BB1FC20, 0x39BECB6A, 0xCF584C4A, 
	0xFBAAEFD0, 0x85334D43, 0x7F02F945, 0xA89F3C50, 
	0x8F40A351, 0xF5389D92, 0x21DAB6BC, 0xD2F3FF10, 
	0xEC130CCD, 0x1744975F, 0x3D7EA7C4, 0x73195D64, 
	0xDC4F8160, 0x88902A22, 0x14B8EE46, 0xDB0B5EDE, 
	0x0A3A32E0, 0x5C240649, 0x62ACD3C2, 0x79E49591, 
	0x6D37C8E7, 0xA94ED58D, 0xEAF4566C, 0x08AE7A65, 
	0x2E2578BA, 0xC6B4A61C, 0x1F74DDE8, 0x8A8BBD4B, 
	0x66B53E70, 0x0EF60348, 0xB9573561, 0x9E1DC186, 
	0x1198F8E1, 0x948ED969, 0xE9871E9B, 0xDF2855CE, 
	0x0D89A18C, 0x6842E6BF, 0x0F2D9941, 0x16BB54B0
};

static __constant__ uint32_t d_AES_FSb[64];

static void aes_reg_cpu_init()
{
	hipMemcpyToSymbol(HIP_SYMBOL( d_AES_FSb),
                        FSb,
                        sizeof(FSb),
                        0, hipMemcpyHostToDevice);
}

static __device__ __forceinline__ void aes_reg_gpu_init(uint32_t *regs)
{
	int idx = (threadIdx.x & 0x1F)<<1;
	regs[0] = d_AES_FSb[idx+0];
	regs[1] = d_AES_FSb[idx+1];
}

// Diese funktionen dienen nur als beispiel und müssen selbst implementiert werden!!!
static __device__ __forceinline__ uint32_t aes_reg_getByIndex(int idx, uint32_t r0, uint32_t r1)
{
	int regThreadIndex = idx>>3;
	int regByteIndex = idx & 0x07;

	uint32_t t0 = __shfl((int)r0, regThreadIndex, 32);
	uint32_t t1 = __shfl((int)r1, regThreadIndex, 32);

	uint32_t x = __byte_perm(t0, t1, regByteIndex) & 0xFF;
	uint32_t y = ( ( x << 1 ) ^ ( ( x & 0x80 ) ? 0x1B : 0x00 ) ) & 0xFF;
    uint32_t z =  ( y ^ x );

	return y | __byte_perm(x, z, 0x4001);

	/*
	return	( y       ) ^
            ( x <<  8 ) ^
            ( x << 16 ) ^
            ( z << 24 );
	*/
}

static __device__ __forceinline__ void aes_reg_round_quad(
	uint32_t r0, uint32_t r1,
	uint32_t x0, uint32_t x1, uint32_t x2, uint32_t x3,
	uint32_t k, uint32_t &y)
{
	uint32_t idx0, idx1, idx2, idx3;
	idx0 = __byte_perm(x0, 0, 0x4440);
	idx1 = __byte_perm(x1, 0, 0x4441);
	idx2 = __byte_perm(x2, 0, 0x4442);
	idx3 = __byte_perm(x3, 0, 0x4443);

	idx0 = aes_reg_getByIndex(idx0, r0, r1);
	idx1 = aes_reg_getByIndex(idx1, r0, r1);
	idx2 = aes_reg_getByIndex(idx2, r0, r1);
	idx3 = aes_reg_getByIndex(idx3, r0, r1);

	y = idx0 ^
		__byte_perm(idx1, 0, 0x2103) ^
		__byte_perm(idx2, 0, 0x1032) ^
		__byte_perm(idx3, 0, 0x0321) ^ k;
}

static __device__ __forceinline__ void aes_reg_round(
	uint32_t r0, uint32_t r1,
	uint32_t x0, uint32_t x1, uint32_t x2, uint32_t x3, 
	uint32_t k0, uint32_t k1, uint32_t k2, uint32_t k3, 
	uint32_t &y0, uint32_t &y1, uint32_t &y2, uint32_t &y3)
{
	uint32_t idx0, idx1, idx2, idx3;

	// 0
	idx0 = __byte_perm(x0, 0, 0x4440);
	idx1 = __byte_perm(x1, 0, 0x4441);
	idx2 = __byte_perm(x2, 0, 0x4442);
	idx3 = __byte_perm(x3, 0, 0x4443);

	idx0 = aes_reg_getByIndex(idx0, r0, r1);
	idx1 = aes_reg_getByIndex(idx1, r0, r1);
	idx2 = aes_reg_getByIndex(idx2, r0, r1);
	idx3 = aes_reg_getByIndex(idx3, r0, r1);

	y0 = idx0 ^
		__byte_perm(idx1, 0, 0x2103) ^
		__byte_perm(idx2, 0, 0x1032) ^
		__byte_perm(idx3, 0, 0x0321) ^ k0;

	// 1
	idx0 = __byte_perm(x1, 0, 0x4440);
	idx1 = __byte_perm(x2, 0, 0x4441);
	idx2 = __byte_perm(x3, 0, 0x4442);
	idx3 = __byte_perm(x0, 0, 0x4443);

	idx0 = aes_reg_getByIndex(idx0, r0, r1);
	idx1 = aes_reg_getByIndex(idx1, r0, r1);
	idx2 = aes_reg_getByIndex(idx2, r0, r1);
	idx3 = aes_reg_getByIndex(idx3, r0, r1);

	y1 = idx0 ^
		__byte_perm(idx1, 0, 0x2103) ^
		__byte_perm(idx2, 0, 0x1032) ^
		__byte_perm(idx3, 0, 0x0321) ^ k1;

	// 2
	idx0 = __byte_perm(x2, 0, 0x4440);
	idx1 = __byte_perm(x3, 0, 0x4441);
	idx2 = __byte_perm(x0, 0, 0x4442);
	idx3 = __byte_perm(x1, 0, 0x4443);

	idx0 = aes_reg_getByIndex(idx0, r0, r1);
	idx1 = aes_reg_getByIndex(idx1, r0, r1);
	idx2 = aes_reg_getByIndex(idx2, r0, r1);
	idx3 = aes_reg_getByIndex(idx3, r0, r1);

	y2 = idx0 ^
		__byte_perm(idx1, 0, 0x2103) ^
		__byte_perm(idx2, 0, 0x1032) ^
		__byte_perm(idx3, 0, 0x0321) ^ k2;


	// 3
	idx0 = __byte_perm(x3, 0, 0x4440);
	idx1 = __byte_perm(x0, 0, 0x4441);
	idx2 = __byte_perm(x1, 0, 0x4442);
	idx3 = __byte_perm(x2, 0, 0x4443);

	idx0 = aes_reg_getByIndex(idx0, r0, r1);
	idx1 = aes_reg_getByIndex(idx1, r0, r1);
	idx2 = aes_reg_getByIndex(idx2, r0, r1);
	idx3 = aes_reg_getByIndex(idx3, r0, r1);

	y3 = idx0 ^
		__byte_perm(idx1, 0, 0x2103) ^
		__byte_perm(idx2, 0, 0x1032) ^
		__byte_perm(idx3, 0, 0x0321) ^ k3;
}

// global memory uint32_t read for streaming operation
__device__ __forceinline__ uint32_t load_global_cs(const uint32_t *addr)
{
    uint32_t value;
    asm("ld.global.cs.u32 %0, [%1];" : "=r"(value) : "l"(addr) );
    return value;
}

// global memory uint32_t write for streaming operation
__device__ __forceinline__ void store_global_cs(const uint32_t *addr, uint32_t val)
{
    asm("st.global.cs.u32 [%0], %1;" :: "l"(addr) "r"(val) );
}

__device__ __forceinline__ int CalculateMemoryPosition(uint32_t *addr)
{
	// calculate the position of the element
	int pos = (int)((addr[0] >> 4) & 0x1ffff);
	// return exact element (in uint32_t, not in bytes)
	// j * AES_BLOCK_SIZE = j * 16 (in bytes) = j * 4 (in uint32_t's)
	
	return pos<<2;
}

// TODO: check the code posted here if it is more efficient
// http://stackoverflow.com/questions/6162140/128-bit-integer-on-cuda

#if AES_WITHOUT_SHARED
__device__ __forceinline__ void EncryptAES_ECB(uint32_t r0, uint32_t r1, const uint32_t *roundKeys, uint32_t *state)
{
#pragma unroll 5
	for(int i=0;i<5;i++)
	{
		int rkOffset = i<<3;
		uint32_t temp[4];

		aes_reg_round(r0, r1,
			state[0], state[1], state[2], state[3],
			roundKeys[0+rkOffset], roundKeys[1+rkOffset], roundKeys[2+rkOffset], roundKeys[3+rkOffset], 
			temp[0], temp[1], temp[2], temp[3]);

		aes_reg_round(r0, r1,
			temp[0], temp[1], temp[2], temp[3],
			roundKeys[4+rkOffset], roundKeys[5+rkOffset], roundKeys[6+rkOffset], roundKeys[7+rkOffset],
			state[0], state[1], state[2], state[3]);
	}
}
#else
__device__ __forceinline__ void EncryptAES_ECB(const uint32_t *sharedMemory, const uint32_t *roundKeys, uint32_t *state)
{
#pragma unroll 5
	for(int i=0;i<5;i++)
	{
		int rkOffset = i<<3;
		uint32_t temp[4];

		aes_round(sharedMemory,
			state[0], state[1], state[2], state[3],
			roundKeys[0+rkOffset], roundKeys[1+rkOffset], roundKeys[2+rkOffset], roundKeys[3+rkOffset], 
			temp[0], temp[1], temp[2], temp[3]);

		aes_round(sharedMemory,
			temp[0], temp[1], temp[2], temp[3],
			roundKeys[4+rkOffset], roundKeys[5+rkOffset], roundKeys[6+rkOffset], roundKeys[7+rkOffset],
			state[0], state[1], state[2], state[3]);
	}
}
#endif


__global__ void cuda_mro_hash_prepare_octo(int threads,
					   uint32_t *g_prepMemory, uint32_t *g_keccakOutputData,
					   uint32_t *g_keccakOutputExpandedAESKey,
					   uint32_t *g_abInput, uint32_t *g_ArrAB)
{
#if AES_WITHOUT_SHARED
	uint32_t aes[2];
	aes_reg_gpu_init(aes);
#else
	__shared__ uint32_t sharedMemory[1024];
	aes_gpu_init_flex(sharedMemory);
#endif
	int thread = (blockDim.x * blockIdx.x + threadIdx.x)>>3;
    if (thread < threads)
	{
		int octId = (threadIdx.x & 0x07);
		////// Keccak is precalculated		

		uint32_t text[4], key[40];
		uint32_t thr34 = thread*34;
		uint32_t thr40 = thread*40;		
		uint32_t thr16 = (thread<<4);
		uint32_t thr32 = (thread<<5);

#if DATA_LAYOUT_SWITCH
		uint32_t thread4 = (thread<<2);
#else
		uint32_t thr2MB = (thread<<19);		
#endif

		// copy text & key
		#pragma unroll 4
		for(int j=0;j<4;j++)
			text[j] = g_keccakOutputData[thr34 + j + (octId<<2)];
		#pragma unroll 40
		for(int j=0;j<40;j++)
			key[j] = g_keccakOutputExpandedAESKey[thr40 + j];

		for(int i=0;i<0x4000;i++) // 16k
		{
			// text will be updated
#if AES_WITHOUT_SHARED
			EncryptAES_ECB(aes[0], aes[1], key, text);
#else
			EncryptAES_ECB(sharedMemory, key, text);			
#endif
			// copy to output memory
			// compute index:
			// AES_BLOCKSIZE = 16, INIT_SIZE_BYTE = 128
			// (j * AES_BLOCKSIZE) + (i * INIT_SIZE_BYTE) ^= Bytes
			// (octId << 4) + (i << 7) ^= Bytes
			// (octId << 2) + (i << 5) ^= uint32_t's						
			
			//int offset = (octId << 2) + (i << 5);
			//#pragma unroll 4
			//for(int j=0;j<4;j++)
				//g_prepMemory[thr2MB + offset + j] = text[j];

			// new Data Layout
			// 16x uint32_t per thread (one AES_BLOCK)
			//  4x uint32_t per octID (one quad of AES_BLOCK)
			// stride = number of threads
			// full size = number of threads * 16
#if DATA_LAYOUT_SWITCH
			int offset = (octId<<2) + (i << 5);
			#pragma unroll 4
			for(int j=0;j<4;j++)
				g_prepMemory[j + (thread4) + (offset*threads) ] = text[j];
#else
			int offset = (octId << 2) + (i << 5);
			#pragma unroll 4
			for(int j=0;j<4;j++)
				g_prepMemory[thr2MB + offset + j] = text[j];
#endif
		}

		/////// Write a + b parallel
		uint32_t t1, t2;
		t1 = g_abInput[thr16 + octId];
		t2 = g_abInput[thr16 + octId + 8];

		uint32_t temp =	t1 ^ t2;
		g_ArrAB[thr32 + octId] = temp;
	}
}

__global__ void cuda_mro_hash_post_octo(int threads, uint32_t *g_prepMemory, 
							uint32_t *g_keccakOutputData, uint32_t *g_keccakOutputExpandedAESKey,
							uint32_t *g_hashOutput)
{
#if AES_WITHOUT_SHARED
	uint32_t aes[2];
	aes_reg_gpu_init(aes);
#else
	__shared__ uint32_t sharedMemory[1024];
	aes_gpu_init_flex(sharedMemory);
#endif

	int thread = (blockDim.x * blockIdx.x + threadIdx.x)>>3;
    if (thread < threads)
	{
		int octId = (threadIdx.x & 0x07);
		uint32_t text[4], key[40];
		uint32_t thr34 = thread*34;
		uint32_t thr40 = thread*40;
		uint32_t thr32 = (thread<<5);		

#if DATA_LAYOUT_SWITCH
		uint32_t thread4 = (thread<<2);
#else
		uint32_t thr2MB = (thread<<19);		
#endif

		// copy text & key
		#pragma unroll 4
		for(int j=0;j<4;j++)
			text[j] = g_keccakOutputData[thr34 + j + (octId<<2)];

		#pragma unroll 40
		for(int j=0;j<40;j++)
			key[j] = g_keccakOutputExpandedAESKey[thr40 + j];

		for(int i=0;i<0x4000;i++) // 16k
		{
			// compute index:
			// AES_BLOCKSIZE = 16, INIT_SIZE_BYTE = 128
			// (j * AES_BLOCKSIZE) + (i * INIT_SIZE_BYTE) ^= Bytes
			// (octId << 4) + (i << 7) ^= Bytes
			// (octId << 2) + (i << 5) ^= uint32_t's
			//int offset = (octId << 2) + (i << 5);
			uint32_t block[4];

			// Load Blockdata
			//#pragma unroll 4
			//for(int j=0;j<4;j++)
			//	block[j] = g_prepMemory[thr2MB + offset + j];
			// new Data Layout			
#if DATA_LAYOUT_SWITCH
			int offset = (octId<<2) + (i << 5);
			#pragma unroll 4
			for(int j=0;j<4;j++)
				block[j] = g_prepMemory[j + (thread4) + (offset*threads) ];
#else
			int offset = (octId << 2) + (i << 5);
			#pragma unroll 4
			for(int j=0;j<4;j++)
				block[j] = g_prepMemory[thr2MB + offset + j];
#endif

			// XOR
#pragma unroll 4
			for(int j=0;j<4;j++)
				text[j] ^= block[j];

			// do aes, text will be updated
#if AES_WITHOUT_SHARED
			EncryptAES_ECB(aes[0], aes[1], key, text);
#else			
			EncryptAES_ECB(sharedMemory, key, text);
#endif
		}

		// Output the Hash
		#pragma unroll 4
		for(int j=0;j<4;j++)
			g_hashOutput[thr32 + (octId << 2) + j] = text[j];
	}
}

__global__ void cuda_mro_hash_doWork_quad(int threads, uint32_t *g_prepMemory, uint32_t *g_ArrAB)
{
#if AES_WITHOUT_SHARED
	uint32_t aes[2];
	aes_reg_gpu_init(aes);
#else
	__shared__ uint32_t sharedMemory[1024];
	aes_gpu_init_flex(sharedMemory);
#endif

	int thread = (blockDim.x * blockIdx.x + threadIdx.x)>>2;
    if (thread < threads)
    {
		uint32_t a, b;
		uint32_t c[4];
		uint32_t d;
		uint32_t thr32 = (thread<<5);
#if DATA_LAYOUT_SWITCH
		uint32_t thread4 = (thread<<2);
#else
		uint32_t thr2MB = (thread<<19);		
#endif
		int quadId = (threadIdx.x & 0x03);
		// read a,b from global memory
		a = g_ArrAB[thr32 + quadId];
		b = g_ArrAB[thr32 + quadId + 4];

		for(int i=0;i<0x80000;i++)
		{
			// calculate position
			uint32_t posBase = __shfl((int)a, 0, 4);
			int position = CalculateMemoryPosition(&posBase);

			// new Data Layout
#if DATA_LAYOUT_SWITCH
//			c[0] = g_prepMemory[quadId + thread4 + (position*threads)];
			c[0] = load_global_cs(&g_prepMemory[quadId + thread4 + (position*threads)]);
#else
//			c[0] = g_prepMemory[thr2MB + position + quadId];
			c[0] = load_global_cs(&g_prepMemory[thr2MB + position + quadId]);
#endif

			// to let this function run correctly, earch quad-thread
			// needs an other order of c.
			// The first thread has  c0, c1, c2, c3
			// The second thread has c1, c2, c3, c0
			// The thrird thread has c2, c3, c0, c1
			// The fourth thread has c3, c0, c1, c2
			c[1] = __shfl((int)c[0], quadId + 1, 4);
			c[2] = __shfl((int)c[0], quadId + 2, 4);
			c[3] = __shfl((int)c[0], quadId + 3, 4);
#if AES_WITHOUT_SHARED
			aes_reg_round_quad(aes[0], aes[1],
				c[0], c[1], c[2], c[3],
				a,
				d);
#else
			aes_round_quad(sharedMemory,
				c[0], c[1], c[2], c[3],
				a,
				d);
#endif

			// XOR
			b ^= d;

			// store
#if DATA_LAYOUT_SWITCH
//			g_prepMemory[quadId + thread4 + (position*threads)] = b;
			store_global_cs(&g_prepMemory[quadId + thread4 + (position*threads)], b);
#else
//			g_prepMemory[thr2MB + position + quadId] = b;
			store_global_cs(&g_prepMemory[thr2MB + position + quadId], b);
#endif

			// search new Position
			posBase = __shfl((int)d, 0, 4);
			position = CalculateMemoryPosition(&posBase);

			// read from this position
#if DATA_LAYOUT_SWITCH
//			b = g_prepMemory[quadId + thread4 + (position*threads)];
			b = load_global_cs(&g_prepMemory[quadId + thread4 + (position*threads)]);
#else
//			b = g_prepMemory[thr2MB + position + quadId];
			b = load_global_cs(&g_prepMemory[thr2MB + position + quadId]);
#endif

			// multiply

			// we want to calculate b * d.			
			uint32_t m1[2], m2[2];
			// b
			m1[0] = __shfl((int)b, 0, 4);
			m1[1] = __shfl((int)b, 1, 4);

			// d
			m2[0] = __shfl((int)d, 0, 4);
			m2[1] = __shfl((int)d, 1, 4);			

			// now, thread 1 has b in m1 and thread 3 has d in m2
			uint64_t n1, n2;
			n1 = *(const uint64_t *)m1;
			n2 = *(const uint64_t *)m2;

			uint64_t pResLo = n1 * n2; // thread 2,3
			uint64_t pResHi = __umul64hi(n1, n2); // thread 0,1
			uint64_t pRes = (quadId & 0x02) ? pResLo : pResHi;

			// pRes to uint32_t
			m1[0] = LOWORD(pRes);
			m1[1] = HIWORD(pRes);

			// shuffle
			m2[0] = __shfl((int)m1[1], quadId-1, 4);

			// save
			c[0] = (quadId & 0x01) ? m2[0] : m1[0];

			uint32_t carry;
			c[0] += a;
			carry = (c[0] < a);
			carry = __shfl((int)carry, quadId - 1, 4);
			c[0] += (quadId & 0x01) ? carry : 0;			

			// store
#if DATA_LAYOUT_SWITCH
//			g_prepMemory[quadId + thread4 + (position*threads)] = c[0];
			store_global_cs(&g_prepMemory[quadId + thread4 + (position*threads)], c[0]);
#else
//			g_prepMemory[thr2MB + position + quadId] = c[0];
			store_global_cs(&g_prepMemory[thr2MB + position + quadId], c[0]);
#endif

			// XOR + move c to b
			a = b ^ c[0];
			b = d;
		}
	}
}

#include <stdio.h>

__host__ int cryptonight_num_smx(int thr_id)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, thr_id);
    return props.multiProcessorCount;
}

// Setup-Funktionen
__host__ void cryptonight_cpu_init(int thr_id, int threads)
{
	hipSetDevice(thr_id);

	aes_cpu_init();
	aes_reg_cpu_init();

	size_t mem = 2 * 1024 * 1024;
	mem = mem * (size_t)threads;
	d_prepMemory[thr_id] = NULL;
	hipMalloc(&d_prepMemory[thr_id], mem);

	d_keccakOutputExpandedAESKey1[thr_id] = NULL;
	hipMalloc(&d_keccakOutputExpandedAESKey1[thr_id], 160 * threads);

	d_keccakOutputExpandedAESKey2[thr_id] = NULL;
	hipMalloc(&d_keccakOutputExpandedAESKey2[thr_id], 160 * threads);

	d_keccakOutputData[thr_id] = NULL;
	hipMalloc(&d_keccakOutputData[thr_id], 136 * threads);

	d_hashOutput[thr_id] = NULL;
	hipMalloc(&d_hashOutput[thr_id], 128 * threads);

	d_abInput[thr_id] = NULL;
	hipMalloc(&d_abInput[thr_id], 64 * threads);

	if (d_prepMemory[thr_id] == NULL || d_keccakOutputExpandedAESKey1[thr_id] == NULL || d_keccakOutputExpandedAESKey2[thr_id] == NULL ||
		d_keccakOutputData[thr_id] == NULL || d_hashOutput[thr_id] == NULL || d_abInput[thr_id] == NULL) {
		fprintf(stderr, "Out of memory!\n");
		exit(1);
	}

}

__host__ void cryptonight_cpu_hash_test_flex(int thr_id, int threads, uint32_t *h_keccakOutputData, 
		uint32_t *h_keccakOutputExpandedAESKey1, uint32_t *h_keccakOutputExpandedAESKey2,
		uint32_t *h_abInput, uint32_t *h_hashOutput, int order, const int threadsperblock)
{
	hipMemcpy(d_keccakOutputExpandedAESKey1[thr_id], h_keccakOutputExpandedAESKey1, 160 * threads, hipMemcpyHostToDevice);
	hipMemcpy(d_keccakOutputExpandedAESKey2[thr_id], h_keccakOutputExpandedAESKey2, 160 * threads, hipMemcpyHostToDevice);
	hipMemcpy(d_keccakOutputData[thr_id], h_keccakOutputData, 136 * threads, hipMemcpyHostToDevice);
	hipMemcpy(d_abInput[thr_id], h_abInput, 64 * threads, hipMemcpyHostToDevice);

#if 0
	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
#endif

	// quad kernel quadruples the amount of threads per block
	dim3 gridQuad((threads + threadsperblock-1)/threadsperblock);
	dim3 blockQuad(4*threadsperblock);

	// the octo kernels use 4*threadsperblock and threads*2 for the grid
	// to achieve an expansion of factor 8
	dim3 gridOcto(((threads*2) + threadsperblock-1)/threadsperblock);
	dim3 blockOcto(4*threadsperblock);

	// Größe des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	// hashOutput will be used as temporary memory here
	cuda_mro_hash_prepare_octo<<<gridOcto, blockOcto, shared_size>>>(threads, d_prepMemory[thr_id], d_keccakOutputData[thr_id], d_keccakOutputExpandedAESKey1[thr_id], d_abInput[thr_id], d_hashOutput[thr_id]);
	MyStreamSynchronize(NULL, order, thr_id);
//	cuda_mro_hash_doWork<<<grid, block, shared_size>>>(threads, d_prepMemory[thr_id], d_hashOutput[thr_id]);
	cuda_mro_hash_doWork_quad<<<gridQuad, blockQuad, shared_size>>>(threads, d_prepMemory[thr_id], d_hashOutput[thr_id]);
	MyStreamSynchronize(NULL, order+1, thr_id);
	cuda_mro_hash_post_octo<<<gridOcto, blockOcto, shared_size>>>(threads, d_prepMemory[thr_id], d_keccakOutputData[thr_id], d_keccakOutputExpandedAESKey2[thr_id], d_hashOutput[thr_id]);
	MyStreamSynchronize(NULL, order+2, thr_id);

	//hipMemcpy(h_hashOutput, d_prepMemory[thr_id], 1<<21, hipMemcpyDeviceToHost);	
	hipMemcpy(h_hashOutput, d_hashOutput[thr_id], 128 * threads, hipMemcpyDeviceToHost);	
}

#include <map>
#include <sys/time.h>
#include <unistd.h>

// Zeitsynchronisations-Routine von cudaminer mit CPU sleep
typedef struct { double value[8]; } tsumarray;
hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id)
{
    hipError_t result = hipSuccess;
    if (situation >= 0)
    {   
        static std::map<int, tsumarray> tsum;

        double a = 0.95, b = 0.05;
        if (tsum.find(situation) == tsum.end()) { a = 0.5; b = 0.5; } // faster initial convergence

        double tsync = 0.0;
        double tsleep = 0.95 * tsum[situation].value[thr_id];
        if (hipStreamQuery(stream) == hipErrorNotReady)
        {
            usleep((useconds_t)(1e6*tsleep));
            struct timeval tv_start, tv_end;
            gettimeofday(&tv_start, NULL);
            result = hipStreamSynchronize(stream);
            gettimeofday(&tv_end, NULL);
            tsync = 1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec);
        }
        if (tsync >= 0) tsum[situation].value[thr_id] = a * tsum[situation].value[thr_id] + b * (tsleep+tsync);
    }
    else
        result = hipStreamSynchronize(stream);
    return result;
}

